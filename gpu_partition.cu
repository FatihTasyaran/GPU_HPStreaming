#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/discard_iterator.h>
#include "/home/users/ftasyaran/thrust/thrust/universal_allocator.h"
#include "/home/users/ftasyaran/thrust/thrust/universal_vector.h"
#include <algorithm>
#include <cstdlib>


__host__ void print_two_sets(thrust::universal_vector<int> set1, thrust::universal_vector<int> set2){

  std::cout << "######SET 1######" <<std::endl; 
  for(int i = 0; i < set1.size(); i++){
    std::cout << set1[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "######SET 1######" <<std::endl;

  std::cout << "######SET 2######" <<std::endl; 
  for(int i = 0; i < set2.size(); i++){
    std::cout << set2[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "######SET 2######" <<std::endl; 
  
}

void gpu_part(int m, int* ord, int* ptrs, int* js, int* pv, int no_parts, double& runtime, int* xpins, int* pins, int n){

  thrust::universal_vector<thrust::universal_vector<int>> p2n(no_parts);
  //for(int i = 0; i < no_parts; i++){
  //p2n[i].resize(1000);
  //}
  
  //PRE-PARTITIONING
  //Put first no_parts vertices to each part to get a comparable set

  double slack = 0;
  int min_part = 0;
  int* pw = new int[no_parts];

  double imbal = 1;
  
  for(int i = 0; i < no_parts; i++){
    pw[i] = 1;
  }

  for(int i = 0; i < no_parts; i++){

    int vertex = ord[i]; //Choose a vertex, simulate streaming setting
    pv[vertex] = i;

    for(int p = ptrs[vertex]; p < ptrs[vertex + 1]; p++){
      int nx = js[p];
      p2n[i].push_back(nx);
    }
    
  }
  
  for(int i = 0; i < no_parts; i++){
    printf("Part %d : ", i);
    for(int j = 0; j < p2n[i].size(); j++){
      printf(" %d ", p2n[i][j]);
    }
    printf("\n");
  }
  
  thrust::discard_iterator<> C_begin, C_end;
  thrust::universal_vector<int> nets;

  double pend1, pstart1, pend2, pstart2, vtotal;

  
  int my_part = 0;
  for(int v = no_parts; v < m; v++){//m
    int vertex = ord[v];
    int my_max = 0;
    
    if(((1.5 * imbal * v) / no_parts) > slack) {
      slack = (1.5 * imbal * v) / no_parts ;
    }
    
    printf("Vertex: %d \n", v);
    
    for(int p = ptrs[vertex]; p < ptrs[vertex + 1]; p++){
      nets.push_back(js[p]);
    } //This could be a pre-processing step
    
    for(int part = 0; part < no_parts; part++){
      
      C_end = thrust::set_intersection(p2n[part].begin(), p2n[part].end(), nets.begin(), nets.end(), C_begin);
      
      int size_intersect = C_end - C_begin;
      
      if(size_intersect > my_max && (pw[part] - pw[min_part] < slack)){
	my_part = part;
      }

      if(size_intersect > my_max)
	my_max = part;

      std::cout << "my_part: " << my_part << " part: " << part <<" slack: " << slack << " size_intersect: " << size_intersect <<std::endl
		<< "pw[part]: " << pw[part] << " my_max: " << my_max << std::endl;;

      print_two_sets(p2n[part], nets);
      
    }
    
    pv[vertex] = my_part;
    pw[my_part] += 1;
    
    if(my_part == min_part){
      for(int k = 0; k < no_parts; k++) {
	if(pw[min_part] > pw[k]) {
	  min_part = k;
	}
      }
    }
    
    for(int p = 0; p < nets.size(); p++){
      p2n[my_part].push_back(nets[p]);
    }
    
    nets.resize(0);
    
    for(int i = 0; i < no_parts; i++){
      thrust::sort(p2n[i].begin(), p2n[i].end());
    }
    
  }

  for(int i = 0; i < 100; i++){
    std::cout << "pv[i]: " << pv[i] << std::endl;
  }
  
}

 

